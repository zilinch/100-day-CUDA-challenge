
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void add(int a,int b,int* c){ *c = a + b; }
int main(){
  int *d_c, h_c;
  hipMalloc(&d_c, sizeof(int));
  add<<<1,1>>>(2,3,d_c);
  hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
  printf("2 + 3 = %d\n", h_c);
  hipFree(d_c);
  return 0;
}