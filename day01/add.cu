#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N){
        C[idx] = A[idx] + B[idx];
    }

}
  

int main(){
    const int N = 1024;
    const int size = N * sizeof(int);

    float *v_A = new float[N];
    float *v_B = new float[N];
    float *v_C = new float[N];

    for (int i = 0; i < N; i++){
        v_A[i] = 1;
        v_B[i] = i + 1.5;
    }
    
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size); // GPU space for A
    hipMalloc((void**)&d_B, size); // GPU space for B
    hipMalloc((void**)&d_C, size); // GPU space for C

    // copy input data from host to device:
    hipMemcpy(d_A, v_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, v_B, size, hipMemcpyHostToDevice);

    int numThreads = 256;
    int numBlocks = (N + numThreads - 1) / numThreads;

    vectorAdd<<<numBlocks, numThreads>>>(d_A, d_B, d_C, N);

    hipMemcpy(v_C, d_C, size, hipMemcpyDeviceToHost);

    for(int i = N-10;i<N;i++){
        std::cout << "C[" << i << "] = " << v_C[i] << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    delete[] v_A;
    delete[] v_B;
    delete[] v_C;

    return 0;

}